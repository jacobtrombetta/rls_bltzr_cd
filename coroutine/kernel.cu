#include "hip/hip_runtime.h"
/**
 * nvcc compiler issue with c++20 coroutines
 */
#include "kernel.h"
#include "coroutine.h"

__global__ void print_from_GPU() {
  printf("Printing from GPU\n");
}

void print() {
  print_from_GPU<<<1,1>>>();
  hipDeviceSynchronize();
}
